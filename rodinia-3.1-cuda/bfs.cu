#include "hip/hip_runtime.h"
/*
  Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

  Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Created by Pawan Harish.
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define MAX_THREADS_PER_BLOCK 512
typedef hipStream_t hipStream_t;

int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

#include "kernel.cu"
#include "kernel2.cu"

void BFSGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	no_of_nodes=0;
	edge_list_size=0;
	BFSGraph( argc, argv);
}

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}
////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void BFSGraph( int argc, char** argv) 
{
	///
	const int N=100;
	const int nStreams=2;
	const int nBlocks=2;
	const int nThreads=2;
	int offset=0;

	hipStream_t* streams=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));
	for(int i=0;i<nStream;i++)
		hipStreamCreate(&(streams[i]));
	////
    char *input_f;
	if(argc!=2){
	Usage(argc, argv);
	exit(0);
	}
	
	input_f = argv[1];
	printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(input_f,"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}

	int source = 0;

	fscanf(fp,"%d",&no_of_nodes);

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	// allocate host memory
	/*
    Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
	bool *h_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_updating_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_graph_visited = (bool*) malloc(sizeof(bool)*no_of_nodes);
    */
    //allocated unified memory (UM) -> using only d_*_nodes pointers
    Node* d_graph_nodes;
    hipMallocManaged(&d_graph_nodes, sizeof(Node)*no_of_nodes);
	
    bool *d_graph_mask;
    hipMallocManaged(&d_graph_mask, sizeof(bool)*no_of_nodes);
	
    bool *d_updating_graph_mask;
    hipMallocManaged(&d_updating_graph_mask, sizeof(bool)*no_of_nodes);
	
    bool *d_graph_visited;
    hipMallocManaged(&d_graph_visited, sizeof(bool)*no_of_nodes);


	int start, edgeno;   
	// initalize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
        // change h_* variables to d_* ones
		d_graph_nodes[i].starting = start;
		d_graph_nodes[i].no_of_edges = edgeno;
		d_graph_mask[i]=false;
		d_updating_graph_mask[i]=false;
		d_graph_visited[i]=false;
	}

	//read the source node from the file
	fscanf(fp,"%d",&source);
	source=0;

	//set the source node as true in the mask (unified memory)
	d_graph_mask[source]=true;
	d_graph_visited[source]=true;

	fscanf(fp,"%d",&edge_list_size);

	int id,cost;

	//Copy the Edge List to device Memory
	int* d_graph_edges;
    hipMallocManaged (&d_graph_edges, sizeof(int)*edge_list_size);
	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		d_graph_edges[i] = id;
        //printf("d_graph_edges[%d]: %d\n", i, d_graph_edges[i]);
	}

	if(fp)
		fclose(fp);    

	printf("Read File\n");
    /*
	//Copy the Node list to device memory
	Node* d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

	//Copy the Edge List to device Memory
	int* d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;

	//Copy the Mask to device memory
	bool* d_graph_mask;
	hipMalloc( (void**) &d_graph_mask, sizeof(bool)*no_of_nodes) ;
	hipMemcpy( d_graph_mask, h_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;

	bool* d_updating_graph_mask;
	hipMalloc( (void**) &d_updating_graph_mask, sizeof(bool)*no_of_nodes) ;
	hipMemcpy( d_updating_graph_mask, h_updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
    
	//Copy the Visited nodes array to device memory
	bool* d_graph_visited;
	hipMalloc( (void**) &d_graph_visited, sizeof(bool)*no_of_nodes) ;
	hipMemcpy( d_graph_visited, h_graph_visited, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
    */
    /*
	// allocate mem for the result on host side
	int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
	for(int i=0;i<no_of_nodes;i++)
		h_cost[i]=-1;
	h_cost[source]=0;
	
	// allocate device memory for result
	int* d_cost;
	hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes);
	hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;
    */
	
    // allocate device memory for result (unified memory)
	int* d_cost;
    hipMallocManaged (&d_cost, sizeof(int)*no_of_nodes);
    for(int i=0;i<no_of_nodes;i++)
		d_cost[i]=-1;
	d_cost[source]=0;


	//make a bool to check if the execution is over (unified memory)
	bool *d_over;
	hipMallocManaged(&d_over, sizeof(bool));

	//hipMalloc( (void**) &d_over, sizeof(bool));

	printf("Using Unified Memory... all hipMallocManaged finished.\n");
	//printf("Copied Everything to GPU memory\n");

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	int k=0;
	//printf("Start traversing the tree\n");
	//bool stop;
	//Call the Kernel untill all the elements of Frontier are not false
	//int last_progress=0;
	int* last_progress;
	hipMallocManaged(&last_progress, sizeof(int)*1);
	last_progress[0] = 0;
	do
	{
        *d_over = false;
		// caching code here
		// caching data here, too
		// hipMemPrefetchAsync( , , , );
		Kernel<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes, last_progress);
		// check if kernel execution generated and error
		
		hipDeviceSynchronize();

		Kernel2<<< grid, threads, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over, no_of_nodes);
		// check if kernel execution generated and error
		hipDeviceSynchronize();
		k++;
        printf("k: %d, d_over: %d\n", k, *d_over);
	}
	while(*d_over);


	printf("Kernel Executed %d times\n",k);

	// copy result from device to host
	//hipMemcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) ;

	//Store the result into a file
	FILE *fpo = fopen("result.txt","w");
	for(int i=0;i<no_of_nodes;i++)
		fprintf(fpo,"%d) cost:%d\n",i,d_cost[i]);
	fclose(fpo);
	printf("Result stored in result.txt\n");


	// cleanup memory
    /*
	free( h_graph_nodes);
	free( h_graph_edges);
	free( h_graph_mask);
	free( h_updating_graph_mask);
	free( h_graph_visited);
	free( h_cost);
    */
	for(int i=0;i<nStream;i++)
		hipStreamDestroy((streams[i]));
	hipFree(d_graph_nodes);
	hipFree(d_graph_edges);
	hipFree(d_graph_mask);
	hipFree(d_updating_graph_mask);
	hipFree(d_graph_visited);
	hipFree(d_cost);
}
